#include "hip/hip_runtime.h"
//
// Created by matteo on 7/18/21.
//
#include "GpuSimulation.cuh"
#include "../common/Utils.hpp"

/* Translate a 2D index to a 1D index */
#define index(x, y) x * device::HEIGHT + y

/* Global variables to store block and grid dim */
/* Thread blocks are squares therefore they'll have dimension BLOCK_DIMxBLOCK_DIM */
#define BLOCK_DIM 10
dim3 dim_block;
dim3 dim_grid;

/* ============================================================================================================== */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Device Variables and Constants ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/* ============================================================================================================== */
namespace device {
  CONSTANT unsigned int WIDTH;
  CONSTANT unsigned int HEIGHT;
  CONSTANT LatticeNode INITIAL_CONFIG;
  CONSTANT float W[Q];
  CONSTANT Vector2D<int> e[Q];
}

/* ============================================================================================================== */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Device-Specific Code and Kernels ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/* ============================================================================================================== */

/* Initialize the device lattice using initial config's values */
__global__ void init_kernel(LatticeNode *lattice, LatticeNode *lattice_t) {
  unsigned x_i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y_i = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned index = index(x_i, y_i);
  lattice[index] = lattice_t[index] = device::INITIAL_CONFIG; // Initialize both lattices

  /* TODO: TO REMOVE
   * Put a spike of f at the center */
  if (x_i > 10 && x_i < 20 && y_i > 10 && y_i < 20) {
    for (float &i : lattice[index].f)
      i += 0.02;
  }
}

__device__ inline unsigned clamp(unsigned val, unsigned l, unsigned h) {
  return min(max(val, l), h);
}

/* Stream the fluid */
__device__ void stream(LatticeNode *lattice, LatticeNode *lattice_t) {
  unsigned int x_i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y_i = blockIdx.y * blockDim.y + threadIdx.y;

  /* Move the fluid to neighbouring sites */
  unsigned x_t, y_t;
  unsigned index, index_t;
  index = index(x_i, y_i);

  /* Stream away fluid on each lattice site */
  for (int i = 0; i < Q; i++) {
    x_t = clamp(x_i + device::e[i].x, 0, device::WIDTH - 1);
    y_t = clamp(y_i + device::e[i].y, 0, device::HEIGHT - 1);
    index_t = index(x_t, y_t);
    lattice_t[index_t].f[i] = lattice[index].f[i];
  }

  /* Handle boundaries */
  if (y_i == device::HEIGHT - 1 || y_i == 0 || x_i == device::WIDTH - 1 ||
      x_i == 0) {
    lattice_t[index] = device::INITIAL_CONFIG;
  }
}

/* Collide the fluid */
__device__ void collide(LatticeNode *lattice) {
  unsigned x_i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned y_i = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned index = x_i * device::HEIGHT + y_i;
  if (index > device::WIDTH * device::HEIGHT) {
    printf("ERROR! Index out of bounds");
  }

  float total_density;
  float f_eq;
  float e_dp_u;
  Vector2D<float> new_u{};
  LatticeNode &cur_node = lattice[index];

  /* Compute the total f of lattice site at position (x, y) */
  total_density = 0.0;
  new_u.x = new_u.y = 0.0;

  for (int i = 0; i < Q; i++) {
    total_density += cur_node.f[i];
    /* Accumulate the f inside each component of flow_velocity */
    new_u.x += device::e[i].x * cur_node.f[i]; // U_{x} component
    new_u.y += device::e[i].y * cur_node.f[i]; // U_{y} component
  }
  /* Compute average to get the actual value of flow_velocity */
  /* "Cast" to 0 if the velocity is negative */
  new_u.x = max(0.0, new_u.x / total_density);
  new_u.y = max(0.0, new_u.y / total_density);

  /* Compute densities at thermal equilibrium */
  /* Equation (8) */
  for (int i = 0; i < Q; i++) {
    e_dp_u = new_u * device::e[i];
    f_eq = total_density * device::W[i] *
           (1 + (3 * e_dp_u) + (4.5f * (e_dp_u * e_dp_u)) -
            (1.5f * new_u.mod_sqr()));
    cur_node.f[i] += D2Q9::OMEGA * (f_eq - cur_node.f[i]);
  }
  cur_node.u = new_u;
}
/* Bounce back fluid on boundaries/obstacles */
__device__ void bounce() {

}

/* Perform a LBM step */
__global__ void step_kernel(LatticeNode *lattice, LatticeNode *lattice_t) {
  collide(lattice);
  __threadfence();
  stream(lattice, lattice_t);
  bounce();
}

/* ============================================================================================================== */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ Class Implementation ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/* ============================================================================================================== */
GpuSimulation::GpuSimulation(unsigned int w, unsigned int h) : Simulation(w,
                                                                          h) {
  SIZE = WIDTH * HEIGHT;
  dim_block = dim3(BLOCK_DIM, BLOCK_DIM);
  dim_grid = dim3(WIDTH / dim_block.x, HEIGHT / dim_block.y);
  /* Allocate space for lattice objects on the device */
  hipMemcpyToSymbol(HIP_SYMBOL(device::WIDTH), &WIDTH, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(device::HEIGHT), &HEIGHT, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(device::e), &D2Q9::e, sizeof(Vector2D<int>) * Q);
  hipMemcpyToSymbol(HIP_SYMBOL(device::W), &D2Q9::W, sizeof(float) * Q);

  hipMalloc(&device_lattice, sizeof(LatticeNode) * SIZE);
  hipMalloc(&device_lattice_t, sizeof(LatticeNode) * SIZE);

  host_lattice = new LatticeNode[SIZE];

  /* Initialize the initial configuration */
  float e_dp_u;
  LatticeNode tmp_init_conf;
  tmp_init_conf.u.x = D2Q9::VELOCITY;
  tmp_init_conf.u.y = 0;
  /* Assign each lattice with the equilibrium f */
  for (int i = 0; i < Q; i++) {
    e_dp_u = tmp_init_conf.u * D2Q9::e[i];
    tmp_init_conf.f[i] =
            D2Q9::W[i] * (1 + (3 * e_dp_u) + (4.5 * (e_dp_u * e_dp_u)) -
                          (1.5 * tmp_init_conf.u.mod_sqr()));
  }

  hipMemcpyToSymbol(HIP_SYMBOL(device::INITIAL_CONFIG), &tmp_init_conf,
                     sizeof(LatticeNode));
  /* Initialize */
  printf("Block: %d, %d, %d, Grid: %d, %d, %d\n", dim_block.x, dim_block.y,
         dim_block.z, dim_grid.x, dim_grid.y,
         dim_grid.z);
  init_kernel<<<dim_grid, dim_block>>>(device_lattice, device_lattice_t);
  hipDeviceSynchronize();
}

GpuSimulation::~GpuSimulation() {
  hipFree(device_lattice);
  hipFree(device_lattice_t);
  delete host_lattice;
}

void GpuSimulation::render(SDL_Texture *screen) {
  hipMemcpy(host_lattice, device_lattice, sizeof(LatticeNode) * SIZE,
             hipMemcpyDeviceToHost);
  void *pixels;
  int pitch;
  Uint32 *dest;
  float b;

  if (SDL_LockTexture(screen, nullptr, &pixels, &pitch) < 0) {
    SDL_LogError(SDL_LOG_CATEGORY_APPLICATION, "Couldn't lock texture: %s\n",
                 SDL_GetError());
  }

  for (int y = 0; y < HEIGHT; y++) {
    dest = (Uint32 *) ((Uint8 *) pixels + y * pitch);
    for (int x = 0; x < WIDTH; x++) {
      b = std::min(host_lattice[x * HEIGHT + y].u.modulus() * 4, 1.0f);
      *(dest + x) = utils::HSBtoRGB(0.5, 1.0, b);
    }
  }
  SDL_UnlockTexture(screen);
}

void GpuSimulation::step() {
  hipDeviceSynchronize();
  step_kernel<<<dim_grid, dim_block>>>(device_lattice, device_lattice_t);
  hipDeviceSynchronize();
  LatticeNode *tmp = device_lattice_t;
  device_lattice_t = device_lattice;
  device_lattice = tmp;
}
